#include "hip/hip_runtime.h"
texture<float4, 2, hipReadModeElementType> input_texture;

extern "C" {

  __global__ void kernel2(float4* pos, uchar4* out, ulong pitch, float offset, int kernel_dim)
{
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

    // write output pixel    
    float4 f = make_float4(x / (float)kernel_dim, y / (float)kernel_dim, offset, 0);
    pos[y * pitch + x] = f;
    out[y * (blockDim.x * gridDim.x) + x] = make_uchar4(255.0 * f.x, 255.0 * f.y, 255.0 * f.z, 255.0 * f.w);
}

  __global__ void kernel_fb(float4* out, ulong out_pitch, uchar4* pbo, float offset)
{
  unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
  unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
  unsigned int kernel_dim = gridDim.x*blockDim.y;
  
  //float2 z = make_float2(2.0 * (float) x / kernel_dim - 1.0, 2.0 * (float) y / kernel_dim - 1.0);

  //float4 f = make_float4((z.x + 1.0) / 2.0, (z.y + 1.0) / 2.0, offset, 0);
  //float4 g = tex2D(input_texture, (z.x + 1.0) / 2.0, (z.y + 1.0) / 2.0);
    float4 f = make_float4(x / (float)kernel_dim, y / (float)kernel_dim, offset, 0);
  out[y * out_pitch + x] = f;
  pbo[y * kernel_dim + x] = make_uchar4(255.0 * f.x, 255.0 * f.y, 255.0 * f.z, 255.0 * f.w);
}

}
