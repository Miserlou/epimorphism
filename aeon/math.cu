#include "hip/hip_runtime.h"
__device__ float2 sq(float2 v0){
  return v0 * v0;
}

__device__ float2 sin(float2 v0){
  float s, c;
  sincosf(v0.x, &s, &c);
  return vec2(s * coshf(v0.y), c * sinhf(v0.y));
}

__device__ float2 cos(float2 v0){
  float s, c;
  sincosf(v0.x, &s, &c);
  return vec2(c * coshf(v0.y), -1.0f * s * sinhf(v0.y));
}

__device__ float2 tan(float2 v0){
  float s, c;
  sincosf(2.0f * v0.x, &s, &c);
  float r = c + coshf(2.0f * v0.y);
  return vec2(s, sinhf(2.0f * v0.y)) / r;
}

__device__ float2 sinh(float2 v0){
  float s, c;
  sincosf(v0.y, &s, &c);
  return vec2(sinhf(v0.x) * c, coshf(v0.x) * s);
}

__device__ float2 cosh(float2 v0){
  float s, c;
  sincosf(v0.y, &s, &c);
  return vec2(coshf(v0.x) * c, sinhf(v0.x) * s);
}

__device__ float2 tanh(float2 v0){
  float s, c;
  sincosf(2.0f * v0.y, &s, &c);
  float r = coshf(2.0f * v0.x) + c;
  return vec2(sinhf(2.0f * v0.x), s) / r;
}

__device__ float2 exp(float2 v0){
  float f = expf(v0.x);
  float s, c;
  sincosf(v0.y, &s, &c);
  return vec2(f * c, f * s);
}

__device__ float2 sqrt(float2 v0){
  return vec2(rint(v0.x), rint(v0.y));
}

__device__ float2 G(float2 v0){
  return vec2((v0.x > 0 ? floorf(v0.x) : -1.0f * floorf(-1.0f * v0.x)), (v0.y > 0 ? floorf(v0.y) : -1.0f * floorf(-1.0f * v0.y)));
  // return vec2(floorf(v0.x), floorf(v0.y));
}

__device__ float2 F(float2 v0){
  return v0 - G(v0);
}

__device__ float2 P(float2 v0, float2 v1){
  return vec2(v0.x * v1.x, v0.y * v1.y);
}

//__device__ float2 n(float2 v0){
//  return noise2(v0.x + v0.y);
//}

__device__ float2 H(float2 v0){
  float2 v1 = vec2(par[32], par[33]);
  return vec2(1.0f - v1.x * v0.x * v0.x + v0.y, v1.y * v0.x);
}

__device__ float2 B(float2 v0){
  float K = par[32];
  float pi = PI;
  float mid = v0.x + K * sinf( pi * (v0.y + 1) ) / pi - 1;
  return vec2(mid, v0.y + mid);
}

__device__ float3 rotate3D(float3 v, float3 axis, float th){
  // compute constants
  float c = cosf(th);
  float s = sinf(th);

  // compute rotation
  float3 res = vec3(0.0f, 0.0f, 0.0f);
  res.x = (1.0f + (1.0f - c) * (axis.x * axis.x - 1.0f)) * v.x +
          (axis.z * s + (1.0f - c) * axis.x * axis.y) * v.y +
          (-1.0f * axis.y * s + (1.0f - c) * axis.x * axis.z) * v.z;

  res.y = (-1.0f * axis.z * s + (1.0f - c) * axis.x * axis.y) * v.x +
          (1.0f + (1.0f - c) * (axis.y * axis.y - 1.0f)) * v.y +
          (axis.x * s + (1.0f - c) * axis.y * axis.z) * v.z;

  res.z = (axis.y * s + (1.0f - c) * axis.x * axis.z) * v.x +
          (-1.0f * axis.x * s + (1.0f - c) * axis.y * axis.z) * v.y +
          (1.0f + (1.0f - c) * (axis.z * axis.z - 1.0f)) * v.z;

  return res;
}
