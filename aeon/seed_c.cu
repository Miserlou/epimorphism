#include "hip/hip_runtime.h"
// EPIMORPH library file
// seed color functions for the seed_wca seed

__device__ float4 simple_color(float2 z, float w){
  // simple coloring function

  float a = 0.0f;

  if(_SEED_C_TH_EFF != 0 && (z.y != 0.0f || z.x != 0.0f)){
    a = atan2f(z.y, z.x) * floorf(8.0f * _SEED_C_TH_EFF) / (2.0f * PI);
  }

  // return HSVtoRGB(vec4(_clock * _MOD_SPEED_COLOR * _GLOBAL_SPEED * 0.1f + a, _COLOR_S, w * _COLOR_V * ((1.0f + sin(3.0f * 2.0f * 3.14f * z.x)) / 2.0f) * ((1.0f + cos(3.0f * 2.0f * 3.14f * z.y)) / 2.0f), 0.0f));
  return HSVtoRGB(vec4(_clock * _MOD_SPEED_COLOR * _GLOBAL_SPEED * 0.1f + a, _COLOR_S, w * _COLOR_V, 0.0f));
}
