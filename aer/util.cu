#include "hip/hip_runtime.h"
__device__ float4 vec4(float x, float y, float z, float w){
  return make_float4(x, y, z, w);
}

__device__ float2 vec2(float x, float y){
  return make_float2(x, y);
}

__device__ float dot(float2 z1, float2 z2){
  return z1.x * z2.x + z1.y * z2.y;
}

__device__ float4 operator*(const float m, const float4 z1){
  return vec4(m * z1.x, m * z1.y, m * z1.z, m * z1.w);
}

__device__ float4 operator+(const float4 z1, const float4 z2){
  return vec4(z1.x + z2.x, z1.y + z2.y, z1.z + z2.z, z1.w + z2.w);
}

__device__ float2 operator*(const float m, const float2 z1){
  return vec2(m * z1.x, m * z1.y);
}

__device__ float2 operator*(const float2 z1, const float m){
  return vec2(m * z1.x, m * z1.y);
}

__device__ float2 operator+(const float2 z1, const float2 z2){
  return vec2(z1.x + z2.x, z1.y + z2.y);
}

__device__ float2 operator-(const float2 z1, const float2 z2){
  return vec2(z1.x - z2.x, z1.y - z2.y);
}

__device__ float2 operator*(const float2 z1, const float2 z2){
  return vec2(z1.x * z2.x - z1.y * z2.y, z1.x * z2.y + z1.y * z2.x);
}

__device__ float2 operator/(const float2 z1, const float2 z2){
  float r = dot(z2, z2);
  return vec2((z1.x * z2.x + z1.y * z2.y) / r, (z1.y * z2.x - z1.x * z2.y) / r);
}

__device__ float2 operator/(const float m, const float2 z2){
  float r = dot(z2, z2);
  return vec2((m * z2.x) / r, (-1.0 * m * z2.y) / r);
}

__device__ float mag(const float2 z1){
  return sqrt(z1.x * z1.x + z1.y * z1.y);
}

__device__ float4 tex2D(texture<float4, 2, hipReadModeElementType> tex, float2 z){
  return tex2D(tex, z.x, z.y);
}

__device__ float4 _gamma3(float4 v, float gamma){
  return vec4(pow(v.x, gamma), pow(v.y, gamma), pow(v.z, gamma), v.w);
}

__device__ float rem(float a, float b){
  float tmp = a / b;
  return b * (tmp - floorf(tmp));
}

__device__ float2 rem(float2 z, float b){
  return vec2(rem(z.x, b), rem(z.y, b));
}



__device__ float2 recover(float2 z){
  if(isnan(z.x))
    z.x = 0;
  if(isnan(z.y))
    z.y = 0; 
  return z;
}


