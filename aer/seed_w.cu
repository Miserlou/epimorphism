#include "hip/hip_runtime.h"
// EPIMORPH library file
// seed shape functions for the seed_wca seed function


__device__ float trans_w(float w){
  // EXCLUDE
  float ep = nextafter(0.0f, -1.0f);
  if(w < _SEED_W_THRESH && w > 0.0f)
    w = 1.0f;
  if(w < 0.0f)
    w = _SEED_W_BASE == 0.0f ? ep : _SEED_W_BASE;
  return w;
}

__device__ float solid(float2 z){
  // solid
  return trans_w(1.0f);
}


__device__ float fade(float2 z){
  // linear l-r gradient
  float w = (z.x + 1.0f) / 2.0f;
  return trans_w(w);
}


__device__ float wave(float2 z){
  // sinousoid
  float w = (2.0f + sin(2.0f * 3.14259f * (z.y + /*inc1*/0.0f))) / 4.0f;
  return trans_w(w);
}


__device__ float circle(float2 z){
  // circle
  float r = len(z);
  float w = nextafter(0.0f, -1.0f);
  if(r > 0.5f - _SEED_W / 2.0f && r  < 0.5f + _SEED_W / 2.0f)
    w = (1.0f - 2.0f * fabsf(r - 0.5f) / _SEED_W);
  return trans_w(w);
}


__device__ float lines_lr(float2 z){
  // parallel vertical lines
  float w = nextafter(0.0f, -1.0f);
  if(z.x > (1.0f - _SEED_W))
    w = (z.x - (1.0f - _SEED_W)) / _SEED_W;
  else if(z.x < -1.0f * (1.0f - _SEED_W))
    w = (-1.0f * (1.0f - _SEED_W) - z.x) / _SEED_W;
  return trans_w(w);
}


__device__ float square_fade(float2 z){
  // radially fading square
  float w = nextafter(0.0f, -1.0f);
  if(z.x < _SEED_W && z.x > -1.0f * _SEED_W && z.y < _SEED_W && z.y > -1.0f * _SEED_W)
    w = min((1.0f - fabsf(z.x) / _SEED_W), (1.0f - fabsf(z.y) / _SEED_W));
  return trans_w(w);
}


__device__ float lines_box(float2 z){
  // 4 lines in a box
  float w = nextafter(0.0f, -1.0f);
  if(z.x > (1.0f - _SEED_W))
    w =  (z.y < 0.0f ? max((z.x - (1.0f - _SEED_W)), (-1.0f * (1.0f - _SEED_W) - z.y)) : max((z.x - (1.0f - _SEED_W)), (z.y - (1.0f - _SEED_W)))) / _SEED_W;
  else if(z.y > (1.0f - _SEED_W))
    w =  (z.x > 0.0f ? (z.y - (1.0f - _SEED_W)) : max((z.y - (1.0f - _SEED_W)), -1.0f * (1.0f - _SEED_W) - z.x)) / _SEED_W;
  else if(z.x < -1.0f * (1.0f - _SEED_W))
    w =  (z.y > 0.0f ? (-1.0f * (1.0f - _SEED_W) - z.x) : max((-1.0f * (1.0f - _SEED_W) - z.y), -1.0f * (1.0f - _SEED_W) - z.x)) / _SEED_W;
  else if(z.y < -1.0f * (1.0f - _SEED_W))
    w =  (z.x < 0.0f ? (-1.0f * (1.0f - _SEED_W) - z.y) : max((-1.0f * (1.0f - _SEED_W) - z.y), (z.x - (1.0f - _SEED_W)))) / _SEED_W;
  return trans_w(w);
}


__device__ float lines_box_stag(float2 z){
  // 4 lines in a box, staggered
  float w = nextafter(0.0f, -1.0f);
  if(z.x > (1.0f - _SEED_W))
    w = (z.x - (1.0f - _SEED_W)) / _SEED_W;
  if(z.y > (1.0f - _SEED_W))
    w = (z.y - (1.0f - _SEED_W)) / _SEED_W;
  if(z.x < -1.0f * (1.0f - _SEED_W))
    w = (-1.0f * (1.0f - _SEED_W) - z.x) / _SEED_W;
  if(z.y < -1.0f * (1.0f - _SEED_W) && z.x < (1.0f - _SEED_W))
    w = (-1.0f * (1.0f - _SEED_W) - z.y) / _SEED_W;
  return trans_w(w);
}


__device__ float lines_inner(float2 z){
  // lines in a cross
  float w = nextafter(0.0f, -1.0f);
  if(fabsf(z.x) < _SEED_W)
    w = (1.0f - fabsf(z.x) / _SEED_W);
  if(fabsf(z.y) < _SEED_W)
    w = fmaxf(1.0f - fabsf(z.x) / _SEED_W, 1.0f - fabsf(z.y) / _SEED_W);
  return trans_w(w);
}


__device__ float anti_grid_fade(float2 z){
  // inverse grid, radially shaded
  float w = nextafter(0.0f, -1.0f);
  z = rem(z * 3.0f, 1.0f);
  if((z.x > 0.5f * (1.0f - _SEED_W) && z.x < 0.5f * (1.0f + _SEED_W)) && (z.y < 0.5f * (1.0f + _SEED_W) && z.y > 0.5f * (1.0f - _SEED_W)))
    w = min((1.0f - 2.0f * fabsf(z.y - 0.5f) / _SEED_W), (1.0f - 2.0f * fabsf(z.x - 0.5f) / _SEED_W));
  return trans_w(w);
}


__device__ float grid_fade(float2 z){
  // grid,
  float w = nextafter(0.0f, -1.0f);
  z = rem(z * 3.0f, 1.0f);
  if((z.x > 0.5f * (1.0f - _SEED_W) && z.x < 0.5f * (1.0f + _SEED_W)))
    w = (1.0f - 2.0f * fabsf(z.x - 0.5f) / _SEED_W);
  if((z.y < 0.5f * (1.0f + _SEED_W) && z.y > 0.5f * (1.0f - _SEED_W)))
    w = fmaxf((1.0f - 2.0f * fabsf(z.x - 0.5f) / _SEED_W), (1.0f - 2.0f * fabsf(z.y - 0.5f) / _SEED_W));
  return trans_w(w);
}


__device__ float ball(float2 z){
  // ball, radially shaded
  float w = nextafter(0.0f, -1.0f);
  float r = len(z);
  if(r < _SEED_W)
    w = 1.0f - r / _SEED_W;
  return trans_w(w);
}
