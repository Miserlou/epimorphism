#include "hip/hip_runtime.h"
// EPIMORPH library file
// complex plane reductions C -> [-1, -1] x [1, 1]


__device__ float2 grid_reduce(float2 z){
  // standard reduction based on the cartesian grid
  return rem(z + vec2(1.0f, 1.0f), 2.0f) - vec2(1.0f, 1.0f);
}


__device__ float2 torus_reduce(float2 z){
  // reduction based on the reflective torus
  z = z + vec2(1.0f, 1.0f);

  z = rem(z, 4.0f);
  if(z.x >= 2.0f)
    z.x = 4.0 - z.x;
  if(z.y >= 2.0f)
    z.y = 4.0 - z.y;

  return z - vec2(1.0f, 1.0f);
}
