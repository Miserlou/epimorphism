#include "hip/hip_runtime.h"

__device__ float4 rg_swizzle(float4 v){
  return vec4(v.y, v.x, v.z, v.w);
}

__device__ float4 gb_swizzle(float4 v){
  return vec4(v.x, v.z, v.y, v.w);
}

__device__ float4 rgb_swizzle(float4 v){
  return vec4(v.y, v.z, v.x, v.w);
}

__device__ float4 rotate(float4 v){
  v = RGBtoHSV(v);
  v.x += _COLOR_DHUE;
  return HSVtoRGB(v);
}
