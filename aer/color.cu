#include "hip/hip_runtime.h"
// EPIMORPH library file
// coloring functions

__device__ float4 rgb_id(float4 v, float2 z_z){
  // identity
  return v;
}


__device__ float4 rg_swizzle(float4 v, float2 z_z){
  // red/green switch
  return vec4(v.y, v.x, v.z, v.w);
}


__device__ float4 gb_swizzle(float4 v, float2 z_z){
  // green/blue switch
  return vec4(v.x, v.z, v.y, v.w);
}


__device__ float4 rgb_swizzle(float4 v, float2 z_z){
  // rotate red/green/blue
  return vec4(v.y, v.z, v.x, v.w);
}


__device__ float4 rotate_hsv(float4 v, float2 z_z){
  // hsv rotation
  v = RGBtoHSV(v);

  float l = len(z_z);
  l = (4.0f * _COLOR_LEN_SC + 1.0f) * l / (l + 4.0f * _COLOR_LEN_SC);

  float a = 0.0f;
  if(_COLOR_TH_EFF != 0 && (z_z.y != 0.0f || z_z.x != 0.0f)){
    a = atan2f(z_z.y, z_z.x);
    if(a < 0.0f)
      a += 2.0f * 3.14159f;
    a *= floorf(8.0f * _COLOR_TH_EFF) / (2.0f * 3.14159f);
  }

  float th =  2.0f * PI * (_COLOR_DHUE + l + a + count * _COLOR_SPEED_TH * _GLOBAL_SPEED / 10.0f);

  v.x += th;

  return HSVtoRGB(v);
}


__device__ float4 rotate_hsls(float4 v, float2 z_z){
  // complex hsls rotation
  // defaults
  v = RGBtoHSLs(v);

  float l = len(z_z);
  l = (4.0f * _COLOR_LEN_SC + 1.0f) * l / (l + 4.0f * _COLOR_LEN_SC);

  float a = 0.0f;
  if(_COLOR_TH_EFF != 0 && (z_z.y != 0.0f || z_z.x != 0.0f)){
    a = atan2f(z_z.y, z_z.x);
    if(a < 0.0f)
      a += 2.0f * 3.14159f;
    a *= floorf(8.0f * _COLOR_TH_EFF) / (2.0f * 3.14159f);
  }

  float th =  2.0f * PI * (_COLOR_DHUE + a + l + count * _COLOR_SPEED_TH * _GLOBAL_SPEED / 10.0f);
  float phi = 2.0f * PI * _COLOR_PHI;
  float psi = 2.0f * PI * _COLOR_PSI;
  float c = cosf(th);
  float s = sinf(th);

  float3 axis = vec3(cosf(psi) * cosf(phi), cosf(psi) * sinf(phi), sinf(psi));

  float3 tmp = vec3(0.0f, 0.0f, 0.0f);

  tmp.x = (1.0f + (1.0f - c) * (axis.x * axis.x - 1.0f)) * v.x +
          (axis.z * s + (1.0f - c) * axis.x * axis.y) * v.y +
          (-1.0f * axis.y * s + (1.0f - c) * axis.x * axis.z) * v.z;

  tmp.y = (-1.0f * axis.z * s + (1.0f - c) * axis.x * axis.y) * v.x +
          (1.0f + (1.0f - c) * (axis.y * axis.y - 1.0f)) * v.y +
          (axis.x * s + (1.0f - c) * axis.y * axis.z) * v.z;

  tmp.z = (axis.y * s + (1.0f - c) * axis.x * axis.z) * v.x +
          (-1.0f * axis.x * s + (1.0f - c) * axis.y * axis.z) * v.y +
          (1.0f + (1.0f - c) * (axis.z * axis.z - 1.0f)) * v.z;

  v = vec4(0.9999 * tmp.x, 0.9999 * tmp.y, 0.9999 * tmp.z, v.w);

  return HSLstoRGB(v);
}

